#include "hip/hip_runtime.h"
#include <stdio.h>

#define SIZE 2048 //define the size of vectors
// CUDA kernel for vetcor addition. 
__global__ void vectorAdd(int* A, int* B, int* C, int n) // n varibale is the size but it is not used here
{
    int i=threadIdx.x+blockIdx.x*blockDim.x;
        C[i]=A[i]+B[i];
}
int main() //main function
{
    int* A,* B,* C; //alloctae some pointers for host (CPU)
    int* d_A,* d_B,* d_C; //allocate pointers for device (GPU)
    int size=SIZE*sizeof(int); // size in bytes for the arrays
    // Alloctae host vectors
    A=(int*)malloc(size);
    B=(int*)malloc(size);
    C=(int*)malloc(size);
    // Alloctae device vectors
    hipMalloc((void**)&d_A,size);
    hipMalloc((void**)&d_B,size);
    hipMalloc((void**)&d_C,size);
    // Initialize vetcor A and B
    for(int i=0; i<SIZE; i++){
        A[i]=i;
        B[i]=SIZE-i;

    }
    // copy vectors from host to device
    hipMemcpy(d_A,A, size, hipMemcpyHostToDevice);
    hipMemcpy(d_B,B, size, hipMemcpyHostToDevice);
    // launch the vectorAdd CUDA kernel
    vectorAdd <<<2, 1024>>> (d_A, d_B, d_C, SIZE);//the kernel is called here

    //copy the result back to CPU
    hipMemcpy(C,d_C,size, hipMemcpyDeviceToHost); 
    printf("execution finished");
    for(int i; i<SIZE; i++){
        printf("%d + %d = %d \n",A[i],B[i],C[i]);
    }
    // free memory
    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);
    free(A);
    free(B);
    free(C);

    // here we determine 1 thread and 1 block, so the code prints block ID 0 and ThreadID 0
    // Syntax: kernel_name <<< number of blocks, number of threads per block >>> ();
    // when we use number of blocks=2 and number of threads per block 8, the block ID=1 is executed first 
    hipDeviceSynchronize();//this make the CPU wait for the GPU to finsh runnning the applictaion before continuing. If you comment this the output might not show.
    return 0; 
}
