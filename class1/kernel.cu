#include "hip/hip_runtime.h"

#include <stdio.h>

// here we have 1 kernel and 1 function

// this is the kernel. 
__global__ void test01()
{
    //print the blocks and the threads IDs
    printf("\n The block ID is %d , the thread ID is %d \n", blockIdx.x,threadIdx.x);
    // here blockIdx takes .x because in cuda the cuda numbers can be distributed across dimensions x, y, z 
}
int main() //main function
{
    test01 <<<1, 100>>> ();//the kernel is called here 
    // here we determine 1 thread and 1 block, so the code prints block ID 0 and ThreadID 0
    // Syntax: kernel_name <<< number of blocks, number of threads per block >>> ();
    // when we use number of blocks=2 and number of threads per block 8, the block ID=1 is executed first 
    hipDeviceSynchronize();//this make the CPU wait for the GPU to finsh runnning the applictaion before continuing. If you comment this the output might not show.
    return 0; 
}
