#include "hip/hip_runtime.h"

#include <stdio.h>

// here we have 1 kernel and 1 function

// this is the kernel. 
__global__ void test01()
{
    //print the blocks and the threads IDs
    // ward = 32 threads, lets assume we  have 128 threads/block (change this in line 19: 128>>>) in our project, so 128/32 wards/block
    int ward_ID_value=0;
    ward_ID_value=threadIdx.x/32; 
    printf("\n The block ID is %d , the thread ID is %d , the ward ID is %d  \n", blockIdx.x,threadIdx.x,ward_ID_value);
    // here blockIdx takes .x because in cuda the cuda numbers can be distributed across dimensions x, y, z 
}
int main() //main function
{
    test01 <<<1, 128>>> ();//the kernel is called here
    //test01 <<<1, 64>>> ();//in this case we have 2 wards
    test01 <<<2, 64>>> ();//in this case we have 2 wards per block, so 4 wards in the GPU
    // here we determine 1 thread and 1 block, so the code prints block ID 0 and ThreadID 0
    // Syntax: kernel_name <<< number of blocks, number of threads per block >>> ();
    // when we use number of blocks=2 and number of threads per block 8, the block ID=1 is executed first 
    hipDeviceSynchronize();//this make the CPU wait for the GPU to finsh runnning the applictaion before continuing. If you comment this the output might not show.
    return 0; 
}
