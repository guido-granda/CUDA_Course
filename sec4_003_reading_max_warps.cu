#include <hip/hip_runtime.h>
#include <stdio.h>

int main() {
    int device;
    hipGetDevice(&device); // Get current CUDA device. Provides the total number of NVIDIA GPUs available
    hipDeviceProp_t prop;

    hipGetDeviceProperties(&prop, device); // we dont iterate here because there is only one GPU
    printf("Max_threads_per_SM  0: %d \n", prop.maxThreadsPerMultiProcessor);
    printf("Max_warps_per_SM    0: %d \n\n\n", (prop.maxThreadsPerMultiProcessor)/32);



    //another method to get this information
    int maxThreadsPerMP = 0;
    hipDeviceGetAttribute(&maxThreadsPerMP, hipDeviceAttributeMaxThreadsPerMultiProcessor, device);// write the attribute name sthat you want to read. chech documentation for details
    printf("Max_threads_per_SM  1: %d   \n", maxThreadsPerMP);
    printf("Max_warps_per_SM    1: %d   \n", maxThreadsPerMP/32);

    return 0;
}
