#include "hip/hip_runtime.h"
#include <stdio.h>
//we dont nned to
int main(){
    int nDevices;

    hipGetDeviceCount(&nDevices);//count of devices of our system

    for(int i=0;i<nDevices;i++){
        hipDeviceProp_t prop;// variable to store the properties
        hipGetDeviceProperties( &prop,i);//read all the available properties
        printf("Device number: %d \n",i);
        printf("Device name: %s \n",prop.name);//porp is a structure thta is why we use "." 
        printf("Memory clock rate (KHZ) : %d \n",prop.memoryClockRate);
        printf("Memory bus width (bits): %d\n",prop.memoryBusWidth);
        printf("Peak memory banwidth (Gbits/s): %f \n",2.0*prop.memoryClockRate*prop.memoryBusWidth/8.0/1.0e6);// this metric is very important and we'll need it when comparing the GPU performance for some applications
        printf("Total global memory: %lu \n",prop.totalGlobalMem);
        printf("Compute capability: %d.%d \n",prop.major,prop.minor);//explained in the second section
        printf("Number of Ss: %d \n",prop.multiProcessorCount);
        printf("Max threads per block: %d \n",prop.maxThreadsPerBlock);
        printf("Max threads dimensions: x= %d, y= %d, z=%d \n",prop.maxThreadsDim[0],prop.maxThreadsDim[1],prop.maxThreadsDim[2]);
        printf("Max grid dimensions: x= %d, y=%d, z=%d \n",prop.maxGridSize[0],prop.maxGridSize[1],prop.maxGridSize[3]);

    }
    return 0;

}