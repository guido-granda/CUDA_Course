// here we use the CUDA API to measure performance
#include <hip/hip_runtime.h>
#include <stdio.h>



#define SIZE 1024*800*1024 //define the size of vectors
// CUDA kernel for vetcor addition. 
__global__ void vectorAdd(int* A, int* B, int* C, int n) // n varibale is the size but it is not used here
{
    int i=threadIdx.x+blockIdx.x*blockDim.x;
        C[i]=A[i]+B[i];
}
int main() //main function
{
    int* A,* B,* C; //alloctae some pointers for host (CPU)
    int* d_A,* d_B,* d_C; //allocate pointers for device (GPU)
    int size=SIZE*sizeof(int); // size in bytes for the arrays
    printf("HELLO 00\n");
    // Alloctae host vectors
    A=(int*)malloc(size);
    B=(int*)malloc(size);
    C=(int*)malloc(size);
    printf("HELLO 01\n");
    // Alloctae device vectors
    hipMalloc((void**)&d_A,size);
    hipMalloc((void**)&d_B,size);
    hipMalloc((void**)&d_C,size);
    printf("HELLO 02\n");

    // Initialize vetcor A and B
    for(int i=0; i<SIZE; i++){
        A[i]=i;
        B[i]=SIZE-i;

    }
    // copy vectors from host to device
    printf("HELLO 03\n");

    hipMemcpy(d_A,A, size, hipMemcpyHostToDevice);
    hipMemcpy(d_B,B, size, hipMemcpyHostToDevice);
    // create events for timing
    hipEvent_t start,stop;
    hipEventCreate(&start);// prepares this variable for use
    hipEventCreate(&stop);// prepares this variable for use


    // launch the vectorAdd CUDA kernel
    hipEventRecord(start);
    vectorAdd <<<1024*432, 1024>>> (d_A, d_B, d_C, SIZE);//the kernel is called here, 1024 is the limit of number of threads per blobk in the GPU
    hipEventRecord(stop);

    //copy the result back to CPU
    hipMemcpy(C,d_C,size, hipMemcpyDeviceToHost);
    
    hipEventSynchronize(stop);
    float miliseconds;
    hipEventElapsedTime(&miliseconds,start,stop);

    printf("execution time %f miliseconds",miliseconds);
    for(int i; i<10; i++){
        printf("%d + %d = %d \n",A[i],B[i],C[i]);
    }
    // free memory
    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);
    free(A);
    free(B);
    free(C);

    // here we determine 1 thread and 1 block, so the code prints block ID 0 and ThreadID 0
    // Syntax: kernel_name <<< number of blocks, number of threads per block >>> ();
    // when we use number of blocks=2 and number of threads per block 8, the block ID=1 is executed first 
    hipDeviceSynchronize();//this make the CPU wait for the GPU to finsh runnning the applictaion before continuing. If you comment this the output might not show.
    return 0; 
}
