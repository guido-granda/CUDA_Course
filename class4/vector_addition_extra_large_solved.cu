// here we use the CUDA API to measure performance
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>



#define TOTAL_SIZE 1024*1024*1024 //define the size of vectors
#define CHUNK_SIZE 1024*1024*128 // Elements per chunk, adjust based on available host memory. Each vector divided into 8 pieces
#define BLOCK_SIZE 1024 // number of threads per block (depend on GPU capabilities)
// CUDA kernel for vetcor addition. 
__global__ void vectorAdd(int* A, int* B, int* C, int chunk_size) // n varibale is the size but it is not used here
{
    int i=threadIdx.x+blockIdx.x*blockDim.x;
    if(i< chunk_size){ // ensure we are insite the bounds 
        C[i]=A[i]+B[i];
    }
}
void random_ints(int* x, int size)
{
    for(int i=0; i<size;i++)
    {
      x[i]=rand()%100;  
    }
}
int main() //main function
{
    int* chunk_a,* chunk_b,* chunk_c; //alloctae some pointers for host chunks (CPU)
    int* d_A,* d_B,* d_C; //allocate pointers for device (GPU)
    size_t chunkSizeBytes= CHUNK_SIZE*sizeof(int); // size in bytes for the arrays chunks

    // Alloctae host vectors
    chunk_a=(int*)malloc(chunkSizeBytes);
    chunk_b=(int*)malloc(chunkSizeBytes);
    chunk_c=(int*)malloc(chunkSizeBytes);
    // Alloctae device vectors
    hipMalloc((void**)&d_A,chunkSizeBytes);
    hipMalloc((void**)&d_B,chunkSizeBytes);
    hipMalloc((void**)&d_C,chunkSizeBytes);
    // calculate the number of blocks for the kernel
    int numBlocks=(CHUNK_SIZE+BLOCK_SIZE-1)/BLOCK_SIZE;
    for(long long offset=0; offset<TOTAL_SIZE;offset+=CHUNK_SIZE){
        int current_chunk_size= (TOTAL_SIZE-offset)<CHUNK_SIZE ? (TOTAL_SIZE-offset):CHUNK_SIZE;
        printf("\n Offset %lld \n",offset);
        random_ints(chunk_a,current_chunk_size);
        random_ints(chunk_b,current_chunk_size);

        hipMemcpy(d_A,chunk_a, current_chunk_size*sizeof(int), hipMemcpyHostToDevice);
        hipMemcpy(d_B,chunk_b, current_chunk_size*sizeof(int), hipMemcpyHostToDevice);


        vectorAdd <<<numBlocks, BLOCK_SIZE>>> (d_A, d_B, d_C,current_chunk_size);//the kernel is called here
    
        hipMemcpy(chunk_c,d_C,current_chunk_size*sizeof(int), hipMemcpyDeviceToHost);
    }
    // copy vectors from host to device
    //copy the result back to CPU

    for(int i; i<10; i++){
        printf("%d + %d = %d \n",chunk_a[i],chunk_b[i],chunk_c[i]);
    }
    // free memory
    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);
    free(chunk_a);
    free(chunk_b);
    free(chunk_c);

    // here we determine 1 thread and 1 block, so the code prints block ID 0 and ThreadID 0
    // Syntax: kernel_name <<< number of blocks, number of threads per block >>> ();
    // when we use number of blocks=2 and number of threads per block 8, the block ID=1 is executed first 
    return 0; 
}
